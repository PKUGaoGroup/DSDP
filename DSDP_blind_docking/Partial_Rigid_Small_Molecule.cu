#include "hip/hip_runtime.h"
#include "Partial_Rigid_Small_Molecule.cuh"

// Modified 2023/08/27: 
// 1. read HETATM type in ligand pdbqt file
// 2. read more than 99 atoms in pdbqt file
// by CW Dong
void PARTIAL_RIGID_SMALL_MOLECULE::Initial_From_PDBQT(const char* file_name)
{
	//�Ƚ������
	atom_numbers = 0;
	frc.clear();
	//crd.clear();//����Ҫ���
	crd_from_pdbqt.clear();
	//origin_crd
	//move_vec
	atomic_number.clear();
	atom_mass.clear();
	charge.clear();
	atom_type.clear();
	pdbqt_tree.torsion_numbers = 0;
	pdbqt_tree.node.clear();
	pdbqt_tree.atom_to_node_serial.clear();
	is_pure_H_freedom.clear();
	vina_tree.torsion_numbers = 0;
	vina_tree.node.clear();
	vina_tree.atom_to_node_serial.clear();
	//num_tor//����Ҫ���


	//��pdbqt�ж����������Ϣ����pdbqtͬ����������˵�pdbqt_tree�ĺ���Ҳ������
	NODE temp_node;//���ڲ���node��,����ֻ��ʼ����λ��
	memset(temp_node.matrix, 0, sizeof(float) * 9);
	temp_node.matrix[0] = 1.f, temp_node.matrix[4] = 1.f, temp_node.matrix[8] = 1.f;
	FILE* in = fopen_safely(file_name, "r");
	char str_line[256];
	char str_segment[256] = { '\0' };
	while (true)
	{
		if (strcmp(str_segment, "BRANCH") != 0)
		{
			char* end_test = fgets(str_line, 256, in);
			if (end_test == NULL)
			{
				break;
			}
		}//if ��֤��ǰ��������branch��ʱ���ٶ���һ�ж�����ת��branch��ز���
		sscanf(str_line, "%s", str_segment);

		if (strcmp(str_segment, "ROOT") == 0)
		{
			// Modified 2023/08/27: no warning now
			while (fgets(str_line, 256, in))
			{
				sscanf(str_line, "%s", str_segment);
				if (strcmp(str_segment, "ENDROOT") == 0)
				{
					is_pure_H_freedom.push_back((atom_numbers));
					break;
				}
				// Modified 2023/08/27: 'HETATM'
				else if (strcmp(str_segment, "ATOM") == 0 || strcmp(str_segment, "HETATM") == 0)
				{
					Read_Atom_Line_In_PDBQT(str_line, crd_from_pdbqt, charge, atom_type);
					atomic_number.push_back(Get_Atomic_Number_From_PDBQT_Atom_Name((char*)&atom_type[atom_numbers]));
					//atom_mass.push_back(bd[0].Element_Mass(atomic_number[atom_numbers]));
					pdbqt_tree.atom_to_node_serial.push_back(-1);//root�е�ԭ�Ӳ�ָ���κ�node
					atom_numbers += 1;
				}
				else
				{
					printf("unexpected line in pdbqt:\n%s\n", str_segment);
					exit(-1);
					//getchar();
				}//if atom
			}//while in root
		}//if root
		else if (strcmp(str_segment, "BRANCH") == 0)
		{
			int root_atom_seiral;
			int branch_atom_serial;
			// Modified 2023/08/27: [6] instead of [8], read more than 99 atoms
			sscanf(&str_line[6], " %d %d", &root_atom_seiral, &branch_atom_serial);
			root_atom_seiral -= 1;//pdbqt��1����ԭ��
			branch_atom_serial -= 1;
			int heavy_atom_numbers = 0;//��¼�ýڵ����ж��ٸ�����ԭ�ӣ������ж��Ƿ����ǻ��༯��
			// Modified 2023/08/27: no warning now
			while (fgets(str_line, 256, in))
			{
				sscanf(str_line, "%s", str_segment);
				if (strcmp(str_segment, "BRANCH") == 0 || strcmp(str_segment, "ENDBRANCH") == 0)
				{
					if (strcmp(str_segment, "ENDBRANCH") == 0
						&& heavy_atom_numbers == 1)
					{
						is_pure_H_freedom.push_back(-(atom_numbers));
					}//�����ǻ�����vina��ʵ�ʲ��������ɶȵĽڵ㣬���Ȼ�������ӽڵ㣬��˱�Ȼ��ENDBRANCH����Ϊpdbqt��tree��������������ȵ�
					else
					{
						is_pure_H_freedom.push_back((atom_numbers));
					}
					break;
				}
				// Modified 2023/08/27: 'HETATM'
				else if (strcmp(str_segment, "ATOM") == 0 || strcmp(str_segment, "HETATM") == 0)
				{
					Read_Atom_Line_In_PDBQT(str_line, crd_from_pdbqt, charge, atom_type);
					atomic_number.push_back(Get_Atomic_Number_From_PDBQT_Atom_Name((char*)&atom_type[atom_numbers]));
					//atom_mass.push_back(bd[0].Element_Mass(atomic_number[atom_numbers]));
					pdbqt_tree.atom_to_node_serial.push_back(pdbqt_tree.torsion_numbers);
					if (atomic_number[atom_numbers] != 1)
					{
						heavy_atom_numbers += 1;
					}
					atom_numbers += 1;
				}
				else
				{
					printf("unexpected line in pdbqt:\n%s\n", str_segment);
					exit(-1);
					//getchar();
				}//if atom
			}//while in a branch

			//����ÿ��branch���������Ը���һ������node��Ϣ
			temp_node.root_atom_serial = root_atom_seiral;
			temp_node.branch_atom_serial = branch_atom_serial;
			temp_node.a0 = crd_from_pdbqt[root_atom_seiral];
			temp_node.n0 = crd_from_pdbqt[branch_atom_serial];
			temp_node.n0.x -= temp_node.a0.x;
			temp_node.n0.y -= temp_node.a0.y;
			temp_node.n0.z -= temp_node.a0.z;
			float temp_length = 1.f / sqrtf(temp_node.n0.x * temp_node.n0.x + temp_node.n0.y * temp_node.n0.y + temp_node.n0.z * temp_node.n0.z);
			temp_node.n0.x *= temp_length;
			temp_node.n0.y *= temp_length;
			temp_node.n0.z *= temp_length;
			temp_node.a0.x -= crd_from_pdbqt[0].x;
			temp_node.a0.y -= crd_from_pdbqt[0].y;
			temp_node.a0.z -= crd_from_pdbqt[0].z;//���ں�����õ�origin crd���õ�һ��ԭ�Ӵ�������ԭ��ģ��������Ҫ��ȥ��һ��ԭ������
			temp_node.a = temp_node.a0;
			temp_node.n = temp_node.n0;
			temp_node.last_node_serial = pdbqt_tree.atom_to_node_serial[root_atom_seiral];
			pdbqt_tree.node.push_back(temp_node);

			pdbqt_tree.torsion_numbers += 1;
		}//else if branch
	}
	fclose(in);

	//��pdbqt tree�򻯵�vina tree
	for (int i = 0; i < is_pure_H_freedom[0]; i = i + 1)
	{
		vina_tree.atom_to_node_serial.push_back(-1);
	}//pdbqt�е�rootҲһ����vina�е�root
	for (int i = 0; i < pdbqt_tree.torsion_numbers; i = i + 1)
	{
		int node_serial;
		if (is_pure_H_freedom[i + 1] < 0)
		{
			node_serial = vina_tree.atom_to_node_serial[pdbqt_tree.node[i].root_atom_serial];
		}//if �Ǵ�����ţ���û����ڵ�ԭ�ӹ�����һ�ڵ�Ľڵ���ţ����ڽڵ����¹�������˲���ֱ��ֻ�ýڵ���Ŷ���Ҫ��root ԭ�������ڵ���Ϣ��vina tree�еģ���
		else
		{
			temp_node = pdbqt_tree.node[i];
			temp_node.last_node_serial = vina_tree.atom_to_node_serial[temp_node.root_atom_serial];
			vina_tree.node.push_back(temp_node);

			node_serial = vina_tree.torsion_numbers;
			vina_tree.torsion_numbers += 1;
		}
		for (int j = abs(is_pure_H_freedom[i]); j < abs(is_pure_H_freedom[i + 1]); j = j + 1)
		{
			vina_tree.atom_to_node_serial.push_back(node_serial);
		}
	}

	num_tor = (float)0.5f * (pdbqt_tree.torsion_numbers + vina_tree.torsion_numbers);

	//���괦��
	origin_crd.resize(atom_numbers);
	move_vec = crd_from_pdbqt[0];
	for (int i = 0; i < atom_numbers; i = i + 1)
	{
		origin_crd[i].x = crd_from_pdbqt[i].x - move_vec.x;
		origin_crd[i].y = crd_from_pdbqt[i].y - move_vec.y;
		origin_crd[i].z = crd_from_pdbqt[i].z - move_vec.z;
	}
	crd = origin_crd;
	frc.resize(atom_numbers);
	memset(&frc[0], 0, sizeof(VECTOR) * atom_numbers);

	vina_gpu.Initial(this, &vina_tree);
	pdbqt_gpu.Initial(this, &pdbqt_tree);

	//MC��vina�������
	vina_gpu.last_accepted_energy = 100.f;
	Build_Inner_Neighbor_List
	(atom_numbers, vina_gpu.h_inner_neighbor_list, crd_from_pdbqt, atomic_number,
		vina_tree.atom_to_node_serial);
	hipMemcpy(vina_gpu.inner_neighbor_list, vina_gpu.h_inner_neighbor_list, sizeof(int)*atom_numbers*atom_numbers, hipMemcpyHostToDevice);
	Build_Vina_Atom(vina_gpu.h_vina_atom, atom_type, crd_from_pdbqt, atomic_number);
	hipMemcpy(vina_gpu.d_vina_atom, vina_gpu.h_vina_atom, sizeof(VINA_ATOM)* atom_numbers, hipMemcpyHostToDevice);
}

void PARTIAL_RIGID_SMALL_MOLECULE::GPU::Initial(PARTIAL_RIGID_SMALL_MOLECULE* mol, const TREE* tree)
{
	partial_rigid_small_molecule = mol;
	atom_numbers = partial_rigid_small_molecule[0].atom_numbers;
	node_numbers = tree[0].torsion_numbers;
	u_freedom = node_numbers + 6;

	//gpu��ԭ�������Ϣ���ڴ����
	if (malloced_atom_numbers >= partial_rigid_small_molecule[0].atom_numbers)
	{
		;
	}
	else
	{
		if (origin_crd != NULL)
		{
			hipFree(origin_crd);
			hipFree(ref_crd);
			hipFree(crd);
			hipFree(last_crd);
			hipFree(frc);
			hipFree(atom_to_node_serial);
			hipFree(inner_neighbor_list);
			free(h_inner_neighbor_list);

			//MC��vina�������
			hipHostFree(h_vina_atom);
			hipFree(d_vina_atom);
		}
		hipMalloc((void**)&origin_crd, sizeof(VECTOR) * atom_numbers);
		hipMalloc((void**)&ref_crd, sizeof(VECTOR) * atom_numbers);
		hipMalloc((void**)&crd, sizeof(VECTOR) * atom_numbers);
		hipMalloc((void**)&last_crd, sizeof(VECTOR) * atom_numbers);
		hipMalloc((void**)&frc, sizeof(VECTOR) * atom_numbers);
		hipMalloc((void**)&atom_to_node_serial, sizeof(int) * atom_numbers);
		hipMalloc((void**)&inner_neighbor_list, sizeof(int) * atom_numbers * atom_numbers);
		h_inner_neighbor_list = (int*)malloc(sizeof(int) * atom_numbers * atom_numbers);

		//MC��vina�������
		hipHostMalloc((void**)&h_vina_atom, sizeof(VINA_ATOM) * atom_numbers);
		hipMalloc((void**)&d_vina_atom, sizeof(VINA_ATOM) * atom_numbers);

		malloced_atom_numbers = atom_numbers;
	}

	//gpu�����ɶ������Ϣ���ڴ����
	if (malloced_u_freedom >= u_freedom)
	{
		;
	}
	else
	{
		if (u_crd != NULL)
		{
			hipHostFree(h_u_crd);
			hipFree(u_crd);
			hipHostFree(h_last_accepted_u_crd);
			hipFree(dU_du_crd);
			hipFree(last_u_crd);
			hipFree(last_dU_du_crd);
		}
		hipHostMalloc((void**)&h_u_crd, sizeof(float) * (u_freedom + 1));
		hipMalloc((void**)&u_crd, sizeof(float) * (u_freedom + 1));
		hipHostMalloc((void**)&h_last_accepted_u_crd, sizeof(float) * u_freedom);
		hipMalloc((void**)&dU_du_crd, sizeof(float) * u_freedom);
		hipMalloc((void**)&last_u_crd, sizeof(float) * u_freedom);
		hipMalloc((void**)&last_dU_du_crd, sizeof(float) * u_freedom);
		malloced_u_freedom = u_freedom;
	}

	//gpu�Ͻڵ���Ϣ���ڴ����
	if (malloced_node_numbers >= node_numbers)
	{
		;
	}
	else
	{
		if (node != NULL)
		{
			hipFree(node);
		}
		hipMalloc((void**)&node, sizeof(NODE) * node_numbers);

		malloced_u_freedom = node_numbers;
	}


	hipMemcpy(origin_crd, &partial_rigid_small_molecule[0].origin_crd[0], sizeof(VECTOR) * atom_numbers, hipMemcpyHostToDevice);
	hipMemcpy(ref_crd, &partial_rigid_small_molecule[0].origin_crd[0], sizeof(VECTOR) * atom_numbers, hipMemcpyHostToDevice);
	hipMemcpy(crd, &partial_rigid_small_molecule[0].crd[0], sizeof(VECTOR) * atom_numbers, hipMemcpyHostToDevice);
	hipMemcpy(last_crd, &partial_rigid_small_molecule[0].crd[0], sizeof(VECTOR) * atom_numbers, hipMemcpyHostToDevice);
	hipMemcpy(frc, &partial_rigid_small_molecule[0].frc[0], sizeof(VECTOR) * atom_numbers, hipMemcpyHostToDevice);
	hipMemcpy(atom_to_node_serial, &tree[0].atom_to_node_serial[0], sizeof(int) * atom_numbers, hipMemcpyHostToDevice);

	//��������ֻ�ǳ�ʼ������������õ�u_crd�������ط����и��£����ȫ��Ϊ0.f
	memset(&h_u_crd[0], 0, sizeof(float) * u_freedom);
	hipMemset(u_crd, 0, sizeof(float) * u_freedom);
	hipMemset(last_u_crd, 0, sizeof(float) * u_freedom);
	memset(h_last_accepted_u_crd, 0, sizeof(float) * u_freedom);
	hipMemset(dU_du_crd, 0, sizeof(float) * u_freedom);
	hipMemset(last_dU_du_crd, 0, sizeof(float) * u_freedom);

	hipMemcpy(node, &tree[0].node[0], sizeof(NODE) * node_numbers, hipMemcpyHostToDevice);

}

void PARTIAL_RIGID_SMALL_MOLECULE::Copy_From_PARTIAL_RIGID_SMALL_MOLECULE(PARTIAL_RIGID_SMALL_MOLECULE* input)
{
	atom_numbers = input[0].atom_numbers;
	frc.resize(atom_numbers);
	memset(&frc[0], 0, sizeof(VECTOR) * atom_numbers);
	crd = input[0].crd;
	crd_from_pdbqt = input[0].crd_from_pdbqt;
	origin_crd = input[0].origin_crd;
	move_vec = input[0].move_vec;
	atomic_number = input[0].atomic_number;
	atom_mass = input[0].atom_mass;
	charge = input[0].charge;
	atom_type = input[0].atom_type;

	pdbqt_tree.atom_to_node_serial = input[0].pdbqt_tree.atom_to_node_serial;
	pdbqt_tree.torsion_numbers = input[0].pdbqt_tree.torsion_numbers;
	pdbqt_tree.node = input[0].pdbqt_tree.node;

	is_pure_H_freedom = input[0].is_pure_H_freedom;

	vina_tree.atom_to_node_serial = input[0].vina_tree.atom_to_node_serial;
	vina_tree.torsion_numbers = input[0].vina_tree.torsion_numbers;
	vina_tree.node = input[0].vina_tree.node;

	num_tor = input[0].num_tor;

	vina_gpu.Initial(this, &vina_tree);
	pdbqt_gpu.Initial(this, &pdbqt_tree);

	//MC��vina�������
	vina_gpu.last_accepted_energy = input[0].vina_gpu.last_accepted_energy;
	hipMemcpy(vina_gpu.inner_neighbor_list, input[0].vina_gpu.h_inner_neighbor_list, sizeof(int) * atom_numbers * atom_numbers, hipMemcpyHostToDevice);
	hipMemcpy(vina_gpu.d_vina_atom, input[0].vina_gpu.h_vina_atom, sizeof(VINA_ATOM) * atom_numbers, hipMemcpyHostToDevice);
}

