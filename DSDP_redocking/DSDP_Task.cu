#include "DSDP_Task.cuh"
void DSDP_TASK::Initial()
{
	if (!is_initialized)
	{
		hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);//hipStreamDefault hipStreamNonBlocking
		hipEventCreateWithFlags(&event, hipEventDisableTiming);//��ʼ��ʱ��hipEventQuery����hipSuccess
		status = DSDP_TASK_STATUS::EMPTY;
		is_initialized = true;
	}
}
bool DSDP_TASK::Is_empty()
{
	if (hipEventQuery(event) == hipSuccess)
	{
		return true;
	}
	else
	{
		return false;
	}
}
hipStream_t DSDP_TASK::Get_Stream()
{
	return stream;
}
void DSDP_TASK::Record_Event()
{
	hipEventRecord(event, stream);
}
void DSDP_TASK::Clear()
{
	if (is_initialized)
	{
		hipStreamDestroy(stream);
		hipEventDestroy(event);
		status = DSDP_TASK_STATUS::NOT_INITIALIZED;
		is_initialized = false;
	}
}
void DSDP_TASK::Assign_Status(const DSDP_TASK_STATUS status)
{
	this->status = status;
}

DSDP_TASK_STATUS DSDP_TASK::Get_Status()
{
	return status;
}
