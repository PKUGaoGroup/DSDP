#include "hip/hip_runtime.h"
#include "common.cuh"
#include "Kernel.cuh"
#include "DSDP_Task.cuh"
#include "Partial_Rigid_Small_Molecule.cuh"
#include "Neighbor_Grid.cuh"
#include "Rigid_Protein.cuh"
#include "Vina_Grid_Force_Field.cuh"
#include "Copy_pdbqt_Format.cuh"
#include "DSDP_Sort.cuh"
#include <time.h>

#define OMP_TIME
#ifdef OMP_TIME
#include <omp.h>
#endif // OMP_TIME
#define NEW_PARSE
#ifdef NEW_PARSE
#include "../include/argparse.hpp"
#endif
std::vector<DSDP_TASK> task;
std::vector<PARTIAL_RIGID_SMALL_MOLECULE> molecule;
RIGID_PROTEIN protein;
NEIGHBOR_GRID nl_grid;
VINA_GRID_FORCE_FIELD vgff;
COPY_pdbqt_FORMAT copy_pdbqt;
DSDP_SORT DSDP_sort;

int main(int argn, char *argv[])
{
	// for most situations, below parameters are suitable, no need for others to change
	// neighbor list related
	const VECTOR neighbor_grid_box_length = {400.f, 400.f, 400.f}; // ��ס����ģ��ԭ�ӵĿռ��С���ͽ��ڱ�����أ�
	const float cutoff = 8.f;									   // �ض�
	const float neighbor_grid_skin = 2.f;

	// interpolation list number in one dimension
	const unsigned int protein_mesh_grid_one_dimension_numbers = 100;

	// a factor to restrain small molecule in searching space
	const float border_strenth = 100.f;

	// some vina force field parameters
	const float omega = 0.292300f * 0.2f; // C_inter/(1+omega*N_rot);
	const float beta = 0.838718f;		  // 600 K, 1/(kb*T)

	// the allowed longest running time in one searching turn
	const float max_allowed_running_time = 60.f; // in sec

	// below parameters can be changed while command line input, but default may be good
	unsigned int stream_numbers = 384; // �ð汾ÿ��stream�Ͷ�Ӧһ����������
	unsigned int search_depth = 40;	   // ÿ���������������Ĵ���

	float box_length = 30.f;			  // another space restrain, manily because of the interpolation space limit, larger will slower(if keep interpolation precision in the same time)
	int max_record_numbers = 2000;		  // only consider top 2000 poses by energy sorting
	float rmsd_similarity_cutoff = 2.f;	  // a parameter to distinguish two different poses
	int desired_saving_pose_numbers = 50; // try to find the best 50 results to save

	double time_begin = omp_get_wtime();
	// file name
	char ligand_name[256];	 // ��������
	char protein_name[256];	 // ��������
	VECTOR box_min, box_max; // ��������

	char out_pdbqt_name[256] = "OUT.pdbqt";
	char out_list_name[256] = "OUT.log";
// �����ⲿ����
#ifndef NEW_PARSE
	int necessary_input_number_record = 0;
	for (int i = 0; i < argn; i = i + 1)
	{
		// below's cmd instruction must be inputed
		if (strcmp(argv[i], "-ligand") == 0)
		{
			i += 1;
			sprintf(ligand_name, "%s", argv[i]);
			necessary_input_number_record += 1;
		}
		else if (strcmp(argv[i], "-protein") == 0)
		{
			i += 1;
			sprintf(protein_name, "%s", argv[i]);
			necessary_input_number_record += 1;
		}

		else if (strcmp(argv[i], "-box_min") == 0)
		{
			i += 1;
			sscanf(argv[i], "%f", &box_min.x);
			i += 1;
			sscanf(argv[i], "%f", &box_min.y);
			i += 1;
			sscanf(argv[i], "%f", &box_min.z);
			necessary_input_number_record += 1;
		}
		else if (strcmp(argv[i], "-box_max") == 0)
		{
			i += 1;
			sscanf(argv[i], "%f", &box_max.x);
			i += 1;
			sscanf(argv[i], "%f", &box_max.y);
			i += 1;
			sscanf(argv[i], "%f", &box_max.z);
			necessary_input_number_record += 1;
		}
		// some options
		else if (strcmp(argv[i], "-out") == 0)
		{
			i += 1;
			sprintf(out_pdbqt_name, "%s", argv[i]);
		}
		else if (strcmp(argv[i], "-log") == 0)
		{
			i += 1;
			sprintf(out_list_name, "%s", argv[i]);
		}
		else if (strcmp(argv[i], "-exhaustiveness") == 0)
		{
			i += 1;
			sscanf(argv[i], "%u", &stream_numbers);
		}
		else if (strcmp(argv[i], "-search_depth") == 0)
		{
			i += 1;
			sscanf(argv[i], "%u", &search_depth);
		}
		else if (strcmp(argv[i], "-top_n") == 0)
		{
			i += 1;
			sscanf(argv[i], "%d", &desired_saving_pose_numbers);
		}
	}
	if (necessary_input_number_record != 4)
	{
		printf("please input correct command\n");
		return -1;
		// getchar();
	}
#endif
#ifdef NEW_PARSE
	/* a new argparsing style */
	argparse::ArgumentParser program("DSDP", "1.0", argparse::default_arguments::help);
	program.add_argument("--ligand").required().help("ligand input PDBQT file").metavar("<ligand.pdbqt>");
	program.add_argument("--protein").required().help("protein input PDBQT file").metavar("<protein.pdbqt>");

	program.add_argument("--box_min").required().help("grid_box min: x y z (Angstrom)").metavar("x y z").nargs(3).scan<'g', float>();
	program.add_argument("--box_max").required().help("grid_box max: x y z (Angstrom)").metavar("x y z").nargs(3).scan<'g', float>();

	program.add_argument("--out").help("ligand poses output").metavar("<*.pdbqt>").default_value(std::string("OUT.pdbqt"));
	program.add_argument("--log").help("docking log file").metavar("<*.log>").default_value(std::string("OUT.log"));
	program.add_argument("--exhaustiveness").help("number of GPU thread (=number of copies)").metavar("N").scan<'i', int>().default_value(384);
	program.add_argument("--search_depth").help("number of searching steps for every copy").metavar("N").scan<'i', int>().default_value(40);
	program.add_argument("--top_n").help("number of desired output poses").metavar("N").scan<'i', int>().default_value(10);
	program.add_description("DSDP: Deep Site and Docking Pose\n"
							" This is the `redocking' program, run with known binding site.\n"
							" More details at https://github.com/PKUGaoGroup/DSDP");

	//  "* Cite this: J. Chem. Inf. Model. 2023, 63, 4355-4363\n"
	//  "             https://doi.org/10.1021/acs.jcim.3c00519"

	try
	{
		program.parse_args(argn, argv);
	}
	catch (const std::runtime_error &err)
	{
		std::cerr << err.what() << std::endl;
		std::cerr << program;
		std::exit(1);
	}

	sscanf(program.get<std::string>("--ligand").c_str(), "%s", ligand_name);
	sscanf(program.get<std::string>("--protein").c_str(), "%s", protein_name);
	/* Box Input */
	auto bmin = program.get<std::vector<float>>("--box_min");
	auto bmax = program.get<std::vector<float>>("--box_max");
	box_min = {bmin[0], bmin[1], bmin[2]};
	box_max = {bmax[0], bmax[1], bmax[2]};

	sscanf(program.get<std::string>("--out").c_str(), "%s", out_pdbqt_name);
	sscanf(program.get<std::string>("--log").c_str(), "%s", out_list_name);

	stream_numbers = program.get<int>("--exhaustiveness");
	search_depth = program.get<int>("--search_depth");
	desired_saving_pose_numbers = program.get<int>("--top_n");

#endif
	// ��ʼ��
	srand((int)time(0));
	hipError_t error = hipSetDeviceFlags(hipDeviceScheduleAuto);

	std::vector<VECTOR> crd_record;
	std::vector<INT_FLOAT> energy_record;

	task.resize(stream_numbers);
	for (int i = 0; i < stream_numbers; i = i + 1)
	{
		task[i].Initial();
	}
	molecule.resize(stream_numbers);
	molecule[0].Initial_From_PDBQT(ligand_name);
	for (int i = 1; i < stream_numbers; i = i + 1)
	{
		molecule[i].Copy_From_PARTIAL_RIGID_SMALL_MOLECULE(&molecule[0]);
	}
	copy_pdbqt.Initial(ligand_name);
	int u_freedom = molecule[0].vina_gpu.u_freedom;
	nl_grid.Initial(neighbor_grid_box_length, cutoff, neighbor_grid_skin);
	vgff.Initial(protein_mesh_grid_one_dimension_numbers, cutoff);
	protein.Initial_Protein_From_PDBQT(protein_name, neighbor_grid_box_length);
	nl_grid.gpu.Put_Atom_Into_Grid_Bucket(protein.atom_numbers, &protein.crd[0]);
	box_min.x += protein.move_vec.x;
	box_min.y += protein.move_vec.y;
	box_min.z += protein.move_vec.z;
	box_max.x += protein.move_vec.x;
	box_max.y += protein.move_vec.y;
	box_max.z += protein.move_vec.z;
	vgff.grid.Calculate_Protein_Potential_Grid(
		box_min, box_length,
		protein.atom_numbers, protein.d_vina_atom,
		nl_grid.grid_length_inverse, nl_grid.grid_dimension, nl_grid.gpu.neighbor_grid_bucket);

	// �����ʼ��
	for (int i = 0; i < stream_numbers; i = i + 1)
	{
		VECTOR rand_vec = {(float)0.5f * rand() / RAND_MAX + 0.25f, (float)0.5f * rand() / RAND_MAX + 0.25f, (float)0.5f * rand() / RAND_MAX + 0.25f};
		for (int j = 0; j < u_freedom - 6; j = j + 1)
		{
			molecule[i].vina_gpu.h_u_crd[j] = 2.f * 3.141592654f * rand() / RAND_MAX;
		}
		VECTOR rand_angle = unifom_rand_Euler_angles();
		molecule[i].vina_gpu.h_u_crd[u_freedom - 3] = rand_angle.z;
		molecule[i].vina_gpu.h_u_crd[u_freedom - 2] = rand_angle.y;
		molecule[i].vina_gpu.h_u_crd[u_freedom - 1] = rand_angle.x;

		molecule[i].vina_gpu.h_u_crd[u_freedom - 6] = (box_min.x + rand_vec.x * (box_max.x - box_min.x));
		molecule[i].vina_gpu.h_u_crd[u_freedom - 5] = (box_min.y + rand_vec.y * (box_max.y - box_min.y));
		molecule[i].vina_gpu.h_u_crd[u_freedom - 4] = (box_min.z + rand_vec.z * (box_max.z - box_min.z));

		molecule[i].vina_gpu.last_accepted_energy = 1000.f;
		task[i].Assign_Status(DSDP_TASK_STATUS::EMPTY);
		memcpy(molecule[i].vina_gpu.h_last_accepted_u_crd, molecule[i].vina_gpu.h_u_crd, sizeof(float) * u_freedom);
	}

	// ����
#ifdef OMP_TIME
	double time_start = omp_get_wtime();
#endif // OMP_TIME
	// std::vector<VECTOR>crd_record;
	// std::vector<INT_FLOAT>energy_record;
	std::vector<int> search_numbers_record(stream_numbers, 0);
	hipDeviceSynchronize();
	while (true)
	{
		bool is_ok_to_break = true;
		for (int i = 0; i < stream_numbers; i = i + 1)
		{
			if (task[i].Is_empty())
			{
				if (task[i].Get_Status() == DSDP_TASK_STATUS::MINIMIZE_STRUCTURE)
				{
					float current_energy = molecule[i].vina_gpu.h_u_crd[u_freedom] / (1.f + omega * molecule[i].num_tor); // vina force field, may be for comparing different molecules with different torsion freedoms
					float probability = expf(fminf(beta * (molecule[i].vina_gpu.last_accepted_energy - current_energy), 0.f));
					if (probability > (float)rand() / RAND_MAX)
					{
						molecule[i].vina_gpu.last_accepted_energy = current_energy;
						energy_record.push_back({(int)energy_record.size(), current_energy});
						for (int j = 0; j < molecule[i].atom_numbers; j = j + 1)
						{
							crd_record.push_back(molecule[i].vina_gpu.h_vina_atom[j].crd);
						}
						memcpy(molecule[i].vina_gpu.h_last_accepted_u_crd, molecule[i].vina_gpu.h_u_crd, sizeof(float) * u_freedom);
					}
					else
					{
						memcpy(molecule[i].vina_gpu.h_u_crd, molecule[i].vina_gpu.h_last_accepted_u_crd, sizeof(float) * u_freedom);
					}
				} // ���н��ջ�ܾ������ж�

				// ����Ŷ�
				int rand_int = rand() % u_freedom;
				if (rand_int < u_freedom - 3)
				{
					if (rand_int < u_freedom - 6)
					{
						molecule[i].vina_gpu.h_u_crd[rand_int] = 2.f * 3.141592654f * ((float)rand() / RAND_MAX);
					}
					else
					{
						molecule[i].vina_gpu.h_u_crd[rand_int] += 1.f * (2.f * ((float)rand() / RAND_MAX) - 1.f);
					}
				}
				else
				{
					VECTOR rand_angle = unifom_rand_Euler_angles();
					molecule[i].vina_gpu.h_u_crd[u_freedom - 3] = rand_angle.z;
					molecule[i].vina_gpu.h_u_crd[u_freedom - 2] = rand_angle.y;
					molecule[i].vina_gpu.h_u_crd[u_freedom - 1] = rand_angle.x;
				}

				search_numbers_record[i] += 1;
				hipMemcpyAsync(molecule[i].vina_gpu.u_crd, molecule[i].vina_gpu.h_u_crd, sizeof(float) * (u_freedom + 1), hipMemcpyHostToDevice, task[i].Get_Stream());
				Optimize_Structure_BB2_Direct_Pair_Device<<<1, 128, sizeof(float) * 23, task[i].Get_Stream()>>> // 23 is a temp array's length, just for GPU computation
					(
						molecule[i].atom_numbers, molecule[i].vina_gpu.inner_neighbor_list, cutoff,
						molecule[i].vina_gpu.atom_to_node_serial,
						molecule[i].vina_gpu.ref_crd, molecule[i].vina_gpu.d_vina_atom, molecule[i].vina_gpu.frc, &molecule[i].vina_gpu.u_crd[u_freedom],
						vgff.grid.texObj_for_kernel, border_strenth /*���ӱ߽�ǿ��*/,
						box_min, box_max, vgff.grid_length_inverse,
						u_freedom, molecule[i].vina_gpu.u_crd, molecule[i].vina_gpu.last_u_crd, molecule[i].vina_gpu.dU_du_crd, molecule[i].vina_gpu.last_dU_du_crd,
						molecule[i].vina_gpu.node_numbers, molecule[i].vina_gpu.node);
				hipMemcpyAsync(molecule[i].vina_gpu.h_u_crd, molecule[i].vina_gpu.u_crd, sizeof(float) * (u_freedom + 1), hipMemcpyDeviceToHost, task[i].Get_Stream());
				hipMemcpyAsync(molecule[i].vina_gpu.h_vina_atom, molecule[i].vina_gpu.d_vina_atom, sizeof(VINA_ATOM) * molecule[i].atom_numbers, hipMemcpyDeviceToHost, task[i].Get_Stream());

				task[i].Assign_Status(DSDP_TASK_STATUS::MINIMIZE_STRUCTURE);
				task[i].Record_Event();
			}
			if (search_numbers_record[i] < search_depth)
			{
				is_ok_to_break = false;
			}
		} // for ÿ��stream����
#ifdef OMP_TIME
		if (omp_get_wtime() - time_start > max_allowed_running_time)
		{
			is_ok_to_break = true;
		}
#endif // OMP_TIME
		if (is_ok_to_break)
		{
			break;
		}
	}
	hipDeviceSynchronize();
#ifdef OMP_TIME

	time_start = omp_get_wtime() - time_start;

	printf("%s\n", ligand_name);
#endif // OMP_TIME

	sort(energy_record.begin(), energy_record.end(), cmp);
	// printf("%f\n", energy_record[0].energy);

	float energy_shift = 0.f;
	VECTOR *molecule_crd = &crd_record[(size_t)energy_record[0].id * (molecule[0].atom_numbers)];
	for (int i = 0; i < molecule[0].atom_numbers; i = i + 1)
	{
		VINA_ATOM atom_j;
		VINA_ATOM atom_i = molecule[0].vina_gpu.h_vina_atom[i];
		atom_i.crd = molecule_crd[i];
		int inner_list_start = i * molecule[0].atom_numbers;
		int inner_numbers = molecule[0].vina_gpu.h_inner_neighbor_list[inner_list_start];
		for (int k = 1; k <= inner_numbers; k = k + 1)
		{
			int j = molecule[0].vina_gpu.h_inner_neighbor_list[inner_list_start + k];
			atom_j = molecule[0].vina_gpu.h_vina_atom[j];
			atom_j.crd = molecule_crd[j];
			float2 temp = Vina_Pair_Interaction(atom_i, atom_j);
			energy_shift += temp.y;
		}
	}
	energy_shift /= (1.f + omega * molecule[0].num_tor);

	VECTOR move_vec = {-protein.move_vec.x, -protein.move_vec.y, -protein.move_vec.z};
	FILE *out_pdbqt = fopen(out_pdbqt_name, "w");
	FILE *out_list = fopen(out_list_name, "w");
	if (!out_pdbqt || !out_list)
	{
		perror("DSDP is unable to open files");
		return -1;
	}
	copy_pdbqt.Append_Frame_To_Opened_pdbqt_standard(out_pdbqt, &crd_record[(size_t)energy_record[0].id * (molecule[0].atom_numbers)], move_vec, 0, energy_record[0].energy - energy_shift);
	fprintf(out_list, "%s %f\n", ligand_name, energy_record[0].energy - energy_shift);
	// ���Ҫȥ���������ǰn������
	DSDP_sort.Sort_Structures(
		molecule[0].atom_numbers, &molecule[0].atomic_number[0],
		std::min(max_record_numbers, (int)energy_record.size()), &crd_record[0], &energy_record[0],
		rmsd_similarity_cutoff, desired_saving_pose_numbers, desired_saving_pose_numbers);
	for (int i = 1; i < DSDP_sort.selected_numbers; i += 1)
	{
		copy_pdbqt.Append_Frame_To_Opened_pdbqt_standard(out_pdbqt, &DSDP_sort.selected_crd[(size_t)i * (molecule[0].atom_numbers)], move_vec, i, energy_record[i].energy - energy_shift);
		fprintf(out_list, "%s %f\n", ligand_name, energy_record[i].energy - energy_shift);
	}

	fclose(out_pdbqt);

	fclose(out_list);

	time_begin = omp_get_wtime() - time_begin;
	printf("Total time %lf s\n", time_begin);
	return 0;
}
