#include "hip/hip_runtime.h"
﻿#include "Kernel.cuh"
__device__ __host__ static void Matrix_Multiply_Vector(VECTOR* __restrict__ c, const float* __restrict__ a, const VECTOR* __restrict__ b)
{
	c[0].x = a[0] * b[0].x + a[1] * b[0].y + a[2] * b[0].z;
	c[0].y = a[3] * b[0].x + a[4] * b[0].y + a[5] * b[0].z;
	c[0].z = a[6] * b[0].x + a[7] * b[0].y + a[8] * b[0].z;
}

__global__ void Optimize_Structure_BB2_Direct_Pair_Device
(
	const int atom_numbers, const int* inner_interaction_list, const float cutoff,
	const int* atom_to_node_serial,
	const VECTOR* ref_crd, VINA_ATOM* vina_atom, VECTOR* frc, float* energy,
	const long long int* protein_mesh, const float box_border_strenth,
	const VECTOR box_min, const VECTOR box_max, const VECTOR protein_mesh_grid_length_inverse,
	const int u_freedom, float* u_crd, float* last_u_crd, float* dU_du_crd, float* last_dU_du_crd,
	const int node_numbers, NODE* node
)
{
	//显卡上的共享存储数组。
	//0 当前能量
	//1 前一帧构象能量
	//2-10 整体转动矩阵
	//11-16 原子受力换算到整体转动的梯度所需的变量
	//17-18,19-20,21-22 为计算三种自由度优化步长的分子分母的存储空间
	//23 当前ligand-ligand的能量，方便外部拆分总能量
	__shared__ float shared_data[24];
	float* rot_matrix = &shared_data[2];
	float* alpha1 = &shared_data[17];
	float* alpha2 = &shared_data[19];
	float* alpha3 = &shared_data[21];
	if (threadIdx.x == 0)
	{
		shared_data[0] = 0.f;
		shared_data[1] = BIG_ENERGY;
		shared_data[23] = 0.f;
	}
	for (int i = threadIdx.x; i < u_freedom; i = i + blockDim.x)
	{
		dU_du_crd[i] = 0.f;
		last_dU_du_crd[i] = 0.f;
		last_u_crd[i] = u_crd[i];
	}


	//进行MAX_OPTIMIZE_STEPS次优化
	__syncthreads();
	for (int opt_i = 0; opt_i < MAX_OPTIMIZE_STEPS; opt_i += 1)
	{


		//计算当前广义坐标下每个节点的转动矩阵（每个二面角自由度）和整体转动矩阵（欧拉角alpha beta gamma）
		for (int i = threadIdx.x; i <= node_numbers; i = i + blockDim.x)
		{
			if (i != node_numbers)
			{
				float temp_matrix_1[9];
				float cosa, sina, cosa_1;
				sincosf(u_crd[i], &sina, &cosa);
				cosa_1 = 1.f - cosa;
				VECTOR temp_n0 = node[i].n0;
				temp_matrix_1[0] = cosa_1 * temp_n0.x * temp_n0.x + cosa;
				temp_matrix_1[1] = cosa_1 * temp_n0.x * temp_n0.y;
				temp_matrix_1[2] = cosa_1 * temp_n0.x * temp_n0.z;
				temp_matrix_1[3] = temp_matrix_1[1];
				temp_matrix_1[4] = cosa_1 * temp_n0.y * temp_n0.y + cosa;
				temp_matrix_1[5] = cosa_1 * temp_n0.y * temp_n0.z;
				temp_matrix_1[6] = temp_matrix_1[2];
				temp_matrix_1[7] = temp_matrix_1[5];
				temp_matrix_1[8] = cosa_1 * temp_n0.z * temp_n0.z + cosa;

				node[i].matrix[0] = temp_matrix_1[0];
				node[i].matrix[1] = temp_matrix_1[1] + sina * temp_n0.z;
				node[i].matrix[2] = temp_matrix_1[2] - sina * temp_n0.y;
				node[i].matrix[3] = temp_matrix_1[3] - sina * temp_n0.z;
				node[i].matrix[4] = temp_matrix_1[4];
				node[i].matrix[5] = temp_matrix_1[5] + sina * temp_n0.x;
				node[i].matrix[6] = temp_matrix_1[6] + sina * temp_n0.y;
				node[i].matrix[7] = temp_matrix_1[7] - sina * temp_n0.x;
				node[i].matrix[8] = temp_matrix_1[8];
			}
			else
			{
				float cos_c;
				float sin_c;
				float cos_b;
				float sin_b;
				float cos_a;
				float sin_a;
				sincosf(u_crd[u_freedom - 3], &sin_c, &cos_c);
				sincosf(u_crd[u_freedom - 2], &sin_b, &cos_b);
				sincosf(u_crd[u_freedom - 1], &sin_a, &cos_a);

				rot_matrix[0] = cos_b * cos_c;
				rot_matrix[1] = cos_b * sin_c;
				rot_matrix[2] = -sin_b;
				rot_matrix[3] = cos_c * sin_a * sin_b - cos_a * sin_c;
				rot_matrix[4] = cos_a * cos_c + sin_a * sin_b * sin_c;
				rot_matrix[5] = cos_b * sin_a;
				rot_matrix[6] = cos_a * cos_c * sin_b + sin_a * sin_c;
				rot_matrix[7] = -cos_c * sin_a + cos_a * sin_b * sin_c;
				rot_matrix[8] = cos_a * cos_b;

				shared_data[11] = cos_b;
				shared_data[12] = sin_b;
				shared_data[13] = cos_a;
				shared_data[14] = sin_a;
				shared_data[15] = rot_matrix[8];//cacb
				shared_data[16] = rot_matrix[5];//cbsa
			}
		}
		__syncthreads();


		//对每个原子，利用上面的转动矩阵和ref_crd计算当前真实的crd
		for (int i = threadIdx.x; i < atom_numbers; i = i + blockDim.x)
		{
			int current_node_id = atom_to_node_serial[i];
			frc[i] = { 0.f,0.f,0.f };//在这里先同时清空原子此时收到的力
			VECTOR temp_crd1 = ref_crd[i];
			VECTOR temp_crd2;
			const VECTOR center = ref_crd[0];
			//对该原子所受影响的所有二面角进行遍历（依次按节点连接顺序回溯）
			while (current_node_id != -1)
			{
				temp_crd2.x = temp_crd1.x - node[current_node_id].a0.x;//进行绕轴转动时，需要将轴平移到坐标原点
				temp_crd2.y = temp_crd1.y - node[current_node_id].a0.y;
				temp_crd2.z = temp_crd1.z - node[current_node_id].a0.z;

				Matrix_Multiply_Vector(&temp_crd1, node[current_node_id].matrix, &temp_crd2);

				temp_crd1.x += node[current_node_id].a0.x;
				temp_crd1.y += node[current_node_id].a0.y;
				temp_crd1.z += node[current_node_id].a0.z;

				current_node_id = node[current_node_id].last_node_serial;
			}
			temp_crd1.x -= center.x;//进行整体转动时，要以0号原子作为坐标原点进行
			temp_crd1.y -= center.y;
			temp_crd1.z -= center.z;
			Matrix_Multiply_Vector(&temp_crd2, rot_matrix, &temp_crd1);
			vina_atom[i].crd.x = temp_crd2.x + u_crd[u_freedom - 6] + center.x;//u_crd[u_freedom - 6]为整体平动坐标的x分量
			vina_atom[i].crd.y = temp_crd2.y + u_crd[u_freedom - 5] + center.y;
			vina_atom[i].crd.z = temp_crd2.z + u_crd[u_freedom - 4] + center.z;
		}
		__syncthreads();


		//在当前原子坐标下更新每个二面角转动轴的方向矢量n和位置a
		for (int node_id = threadIdx.x; node_id < node_numbers; node_id = node_id + blockDim.x)
		{
			float temp_length;
			VECTOR tempa, tempn;
			tempa = { vina_atom[node[node_id].root_atom_serial].crd.x,vina_atom[node[node_id].root_atom_serial].crd.y,vina_atom[node[node_id].root_atom_serial].crd.z };
			tempn = { vina_atom[node[node_id].branch_atom_serial].crd.x,vina_atom[node[node_id].branch_atom_serial].crd.y,vina_atom[node[node_id].branch_atom_serial].crd.z };
			tempn.x -= tempa.x;
			tempn.y -= tempa.y;
			tempn.z -= tempa.z;
			temp_length = rnorm3df(tempn.x, tempn.y, tempn.z);
			tempn.x *= temp_length;
			tempn.y *= temp_length;
			tempn.z *= temp_length;
			node[node_id].n = tempn;
			node[node_id].a = tempa;
		}


		//对每个原子计算打分和梯度
		float total_energy_in_thread = 0.f;
		float intra_energy_in_thread = 0.f;
		for (int i = threadIdx.x; i < atom_numbers; i = i + blockDim.x)
		{
			VINA_ATOM atom_j;
			VECTOR temp_force;
			float rij, frc_abs, rij_inverse;
			float4 ans;
			int inner_list_start;
			VINA_ATOM atom_i = vina_atom[i];
			VECTOR force_i = { 0.f,0.f,0.f };
			VECTOR dr;
			if (atom_i.atom_type < HYDROGEN_ATOM_TYPE_SERIAL)//只计算非H原子所受的插值边界与蛋白作用
			{
				//边界墙壁的作用
				dr.x = fdimf(box_min.x, atom_i.crd.x);
				dr.y = fdimf(box_min.y, atom_i.crd.y);
				dr.z = fdimf(box_min.z, atom_i.crd.z);
				force_i.x += box_border_strenth * dr.x;
				force_i.y += box_border_strenth * dr.y;
				force_i.z += box_border_strenth * dr.z;
				total_energy_in_thread += 0.5f * box_border_strenth * (dr.x * dr.x + dr.y * dr.y + dr.z * dr.z);

				dr.x = fdimf(atom_i.crd.x, box_max.x);
				dr.y = fdimf(atom_i.crd.y, box_max.y);
				dr.z = fdimf(atom_i.crd.z, box_max.z);
				force_i.x -= box_border_strenth * dr.x;
				force_i.y -= box_border_strenth * dr.y;
				force_i.z -= box_border_strenth * dr.z;
				total_energy_in_thread += 0.5f * box_border_strenth * (dr.x * dr.x + dr.y * dr.y + dr.z * dr.z);

				//ligand-蛋白相互作用
				VECTOR serial;//ligand原子所处插值格子的分数坐标（加上插值格子的编号）
				serial.x = (atom_i.crd.x - box_min.x) * protein_mesh_grid_length_inverse.x;
				serial.y = (atom_i.crd.y - box_min.y) * protein_mesh_grid_length_inverse.y;
				serial.z = (atom_i.crd.z - box_min.z) * protein_mesh_grid_length_inverse.z;
				//获取位于该位置的原子（atom_type）的打分和梯度
				ans = tex3D<float4>(protein_mesh[atom_i.atom_type], serial.x + 0.5f, serial.y + 0.5f, serial.z + 0.5f);

				total_energy_in_thread += ans.w;
				force_i.x += ans.x;
				force_i.y += ans.y;
				force_i.z += ans.z;
			}

			//计算ligand-ligand相互作用
			inner_list_start = i * atom_numbers;
			int inner_numbers = inner_interaction_list[inner_list_start];
			for (int k = 1; k <= inner_numbers; k = k + 1)
			{
				int j = inner_interaction_list[inner_list_start + k];
				atom_j = vina_atom[j];
				dr = { atom_i.crd.x - atom_j.crd.x, atom_i.crd.y - atom_j.crd.y, atom_i.crd.z - atom_j.crd.z };
				rij = norm3df(dr.x, dr.y, dr.z);
				if (rij < cutoff)
				{
					float surface_distance = rij - atom_i.radius - atom_j.radius;
					float temp_record;

					temp_record = k_gauss1 * expf(-k_gauss1_2 * surface_distance * surface_distance);
					total_energy_in_thread += temp_record;
					intra_energy_in_thread += temp_record;
					frc_abs = 2.f * k_gauss1_2 * temp_record * surface_distance;

					float dp = surface_distance - k_gauss2_c;
					temp_record = k_gauss2 * expf(-k_gauss2_2 * dp * dp);
					total_energy_in_thread += temp_record;
					intra_energy_in_thread += temp_record;
					frc_abs += 2.f * k_gauss2_2 * temp_record * dp;

					temp_record = k_repulsion * surface_distance * signbit(surface_distance);
					total_energy_in_thread += temp_record * surface_distance;
					intra_energy_in_thread += temp_record * surface_distance;
					frc_abs += -2.f * temp_record;

					if ((atom_i.is_hydrophobic & atom_j.is_hydrophobic))
					{
						temp_record = 1.f * k_hydrophobic;
						total_energy_in_thread += temp_record * (k_hydrophobic_ua * signbit(surface_distance - k_hydrophobic_a) + k_hydrophobic_ub * signbit(k_hydrophobic_b - surface_distance) + (((k_hydrophobic_ub - k_hydrophobic_ua) / (k_hydrophobic_b - k_hydrophobic_a)) * (surface_distance - k_hydrophobic_a) + k_hydrophobic_ua) * signbit(k_hydrophobic_a - surface_distance) * signbit(surface_distance - k_hydrophobic_b));
						intra_energy_in_thread += temp_record * (k_hydrophobic_ua * signbit(surface_distance - k_hydrophobic_a) + k_hydrophobic_ub * signbit(k_hydrophobic_b - surface_distance) + (((k_hydrophobic_ub - k_hydrophobic_ua) / (k_hydrophobic_b - k_hydrophobic_a)) * (surface_distance - k_hydrophobic_a) + k_hydrophobic_ua) * signbit(k_hydrophobic_a - surface_distance) * signbit(surface_distance - k_hydrophobic_b));
						frc_abs += -temp_record * ((k_hydrophobic_ub - k_hydrophobic_ua) / (k_hydrophobic_b - k_hydrophobic_a)) * signbit(k_hydrophobic_a - surface_distance) * signbit(surface_distance - k_hydrophobic_b);
					}

					if (((atom_i.is_donor & atom_j.is_acceptor) | (atom_i.is_acceptor & atom_j.is_donor)))
					{
						temp_record = 1.f * k_h_bond;
						total_energy_in_thread += temp_record * (k_h_bond_ua * signbit(surface_distance - k_h_bond_a) + k_h_bond_ub * signbit(k_h_bond_b - surface_distance) + (((k_h_bond_ub - k_h_bond_ua) / (k_h_bond_b - k_h_bond_a)) * (surface_distance - k_h_bond_a) + k_h_bond_ua) * signbit(k_h_bond_a - surface_distance) * signbit(surface_distance - k_h_bond_b));
						intra_energy_in_thread += temp_record * (k_h_bond_ua * signbit(surface_distance - k_h_bond_a) + k_h_bond_ub * signbit(k_h_bond_b - surface_distance) + (((k_h_bond_ub - k_h_bond_ua) / (k_h_bond_b - k_h_bond_a)) * (surface_distance - k_h_bond_a) + k_h_bond_ua) * signbit(k_h_bond_a - surface_distance) * signbit(surface_distance - k_h_bond_b));
						frc_abs += -temp_record * ((k_h_bond_ub - k_h_bond_ua) / (k_h_bond_b - k_h_bond_a)) * signbit(k_h_bond_a - surface_distance) * signbit(surface_distance - k_h_bond_b);
					}

					rij_inverse = 1.f / (rij + 10.e-6f);
					frc_abs *= rij_inverse;
					temp_force.x = frc_abs * dr.x;
					temp_force.y = frc_abs * dr.y;
					temp_force.z = frc_abs * dr.z;
					force_i.x += temp_force.x;
					force_i.y += temp_force.y;
					force_i.z += temp_force.z;
					atomicAdd(&frc[j].x, -temp_force.x);
					atomicAdd(&frc[j].y, -temp_force.y);
					atomicAdd(&frc[j].z, -temp_force.z);
				}
			}
			atomicAdd(&frc[i].x, force_i.x);
			atomicAdd(&frc[i].y, force_i.y);
			atomicAdd(&frc[i].z, force_i.z);
		}

		//合并整体能量，以及ligand-ligand能量
		atomicAdd(&shared_data[0], total_energy_in_thread);
		atomicAdd(&shared_data[23], intra_energy_in_thread);
		__syncthreads();


		//将能量转存到外部GPU地址
		//进行一次涉及到求和的变量的清空（能量，BB的步长（分子、分母））
		if (threadIdx.x == 0)
		{
			energy[0] = shared_data[0];
			energy[1] = shared_data[23];

			//shared_data[1] = shared_data[0];//当要通过前后两帧的能量差来判断收敛时有用
			
			shared_data[0] = 0.f;
			shared_data[23] = 0.f;

			alpha1[0] = 0.f;
			alpha1[1] = 0.f;
			alpha2[0] = 0.f;
			alpha2[1] = 0.f;
			alpha3[0] = 0.f;
			alpha3[1] = 0.f;
		}

		//将每个原子的力加到广义坐标对应的梯度上
		for (int i = threadIdx.x; i < atom_numbers; i = i + blockDim.x)
		{
			VECTOR center = { vina_atom[0].crd.x ,vina_atom[0].crd.y , vina_atom[0].crd.z };
			VECTOR temp_crd2 = { vina_atom[i].crd.x ,vina_atom[i].crd.y , vina_atom[i].crd.z };
			VECTOR temp_crd = temp_crd2;
			VECTOR temp_frc = frc[i];
			VECTOR cross;
			VECTOR rot_axis;

			temp_crd.x = temp_crd2.x - center.x;
			temp_crd.y = temp_crd2.y - center.y;
			temp_crd.z = temp_crd2.z - center.z;

			//整体转动的梯度
			atomicAdd(&dU_du_crd[u_freedom - 1], (temp_frc.y * temp_crd.z - temp_frc.z * temp_crd.y));
			atomicAdd(&dU_du_crd[u_freedom - 2], (-temp_frc.x * (temp_crd.y * shared_data[14] + temp_crd.z * shared_data[13]) + temp_frc.y * temp_crd.x * shared_data[14] + temp_frc.z * temp_crd.x * shared_data[13]));
			atomicAdd(&dU_du_crd[u_freedom - 3], (temp_frc.x * (temp_crd.y * shared_data[15] - temp_crd.z * shared_data[16]) - temp_frc.y * (temp_crd.x * shared_data[15] + temp_crd.z * shared_data[12]) + temp_frc.z * (temp_crd.x * shared_data[16] + temp_crd.y * shared_data[12])));

			//整体平动的梯度
			atomicAdd(&dU_du_crd[u_freedom - 6], temp_frc.x);
			atomicAdd(&dU_du_crd[u_freedom - 5], temp_frc.y);
			atomicAdd(&dU_du_crd[u_freedom - 4], temp_frc.z);

			int current_node_id = atom_to_node_serial[i];
			while (current_node_id != -1)
			{
				temp_crd.x = temp_crd2.x - node[current_node_id].a.x;
				temp_crd.y = temp_crd2.y - node[current_node_id].a.y;
				temp_crd.z = temp_crd2.z - node[current_node_id].a.z;
				rot_axis = node[current_node_id].n;

				cross.x = temp_crd.y * rot_axis.z - temp_crd.z * rot_axis.y;
				cross.y = temp_crd.z * rot_axis.x - temp_crd.x * rot_axis.z;
				cross.z = temp_crd.x * rot_axis.y - temp_crd.y * rot_axis.x;

				atomicAdd(&dU_du_crd[current_node_id], (temp_frc.x * cross.x + temp_frc.y * cross.y + temp_frc.z * cross.z));
				current_node_id = node[current_node_id].last_node_serial;
			}
		}
		__syncthreads();

		//使用BB进行u_crd优化（梯度为dU_du_crd）
		for (int i = threadIdx.x; i < u_freedom; i = i + blockDim.x)
		{
			float s = u_crd[i] - last_u_crd[i];
			float y = dU_du_crd[i] - last_dU_du_crd[i];
			last_u_crd[i] = u_crd[i];
			last_dU_du_crd[i] = dU_du_crd[i];
			if (i < u_freedom - 6)
			{
				atomicAdd(&alpha1[0], y * s);
				atomicAdd(&alpha1[1], y * y);
			}
			else if (i < u_freedom - 3)
			{
				atomicAdd(&alpha2[0], y * s);
				atomicAdd(&alpha2[1], y * y);
			}
			else
			{
				atomicAdd(&alpha3[0], y * s);
				atomicAdd(&alpha3[1], y * y);
			}
		}
		__syncthreads();
		for (int i = threadIdx.x; i < u_freedom; i = i + blockDim.x)
		{
			float du;
			if (i < u_freedom - 6)
			{
				float temp_alpha = fabsf(alpha1[0]) / fmaxf(alpha1[1], 1.e-6f);
				du = temp_alpha * dU_du_crd[i];
				du = copysignf(fmaxf(fminf(fabsf(du), 2.f * 2.f * 3.141592654f), 2.f * 3.141592654f / 100000.f), du);
			}
			else if (i < u_freedom - 3)
			{
				float temp_alpha = fabsf(alpha2[0]) / fmaxf(alpha2[1], 1.e-6f);
				du = temp_alpha * dU_du_crd[i];
				du = copysignf(fmaxf(fabsf(du), 1.f / 10000.f), du);
			}
			else
			{
				float temp_alpha = fabsf(alpha3[0]) / fmaxf(alpha3[1], 1.e-6f);
				du = temp_alpha * dU_du_crd[i];
				du = copysignf(fmaxf(fabsf(du), 2.f * 3.141592654f / 100000.f), du);
			}
			dU_du_crd[i] = 0.f;
			u_crd[i] += du;
		}
		__syncthreads();
	}
}
